#include "hip/hip_runtime.h"
#include "PostProcessingAlgorithms.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

void CheckCudaErrors(hipError_t error) {
	printf("\nCUDA ERROR --- %d (NULL IT'S OK) \n", error);
}

struct CUDASURFACEDATA {

	hipSurfaceObject_t surface;
	hipMipmappedArray_t cuMipArray;
	hipExternalMemory_t cudaExternalMemory;

	void CLEAR() {
		hipFreeMipmappedArray(cuMipArray);
		hipDestroyExternalMemory(cudaExternalMemory);
		hipDestroySurfaceObject(surface);
	}
};

std::unordered_map<ID3D12Resource*, CUDASURFACEDATA> surfacesMap;


__global__ void invertColorsKernel(hipSurfaceObject_t surface, unsigned int width, unsigned int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;
	float4 pixel;

	surf2Dread(&pixel, surface, x * sizeof(float4), y);

	pixel.x = 1.0f - pixel.x;
	pixel.y = 1.0f - pixel.y;
	pixel.z = 1.0f - pixel.z;

	surf2Dwrite(pixel, surface, x * sizeof(float4), y );
}


CUDASURFACEDATA CreateSurfaceObject(ID3D12Resource* texture, ID3D12Device* device) {
	CUDASURFACEDATA data{};

	D3D12_RESOURCE_DESC texDesc = texture->GetDesc();
	HANDLE sharedHandle{};
	SECURITY_ATTRIBUTES secAttr{};
	LPCWSTR name{};
	hr = device->CreateSharedHandle(texture, &secAttr, GENERIC_ALL, name, &sharedHandle);
	const auto texAllocInfo = device->GetResourceAllocationInfo(NULL, 1, &texDesc);

	hipExternalMemoryHandleDesc cuExtmemHandleDesc{};
	cuExtmemHandleDesc.type = hipExternalMemoryHandleTypeD3D12Heap;
	cuExtmemHandleDesc.handle.win32.handle = sharedHandle;
	cuExtmemHandleDesc.size = texAllocInfo.SizeInBytes;
	cuExtmemHandleDesc.flags = hipExternalMemoryDedicated;
	CheckCudaErrors(hipImportExternalMemory(&data.cudaExternalMemory, &cuExtmemHandleDesc));

	cudaExternalMemoryMipmappedArrayDesc cuExtmemMipDesc{};
	cuExtmemMipDesc.extent = make_hipExtent(texDesc.Width, texDesc.Height, texDesc.DepthOrArraySize);
	cuExtmemMipDesc.formatDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	cuExtmemMipDesc.numLevels = 1;

	CheckCudaErrors(cudaExternalMemoryGetMappedMipmappedArray(&data.cuMipArray, data.cudaExternalMemory, &cuExtmemMipDesc));

	hipArray_t cuArray{};
	CheckCudaErrors(hipGetMipmappedArrayLevel(&cuArray, data.cuMipArray, 0));

	hipSurfaceObject_t cuSurface{};
	hipResourceDesc cuResDesc{};
	cuResDesc.resType = hipResourceTypeArray;
	cuResDesc.res.array.array = cuArray;
	CheckCudaErrors(hipCreateSurfaceObject(&data.surface, &cuResDesc));

	return data;
}

void InverseTexture(ID3D12Resource* texture, ID3D12Device* device)
{
	if (!FindInMap(texture)) {
		PRINTF_FDW("CANT FIND TEXTURE SURFACE /// CALL INIT TO MAP");
		InitToMap(texture, device);
	}

	D3D12_RESOURCE_DESC texDesc = texture->GetDesc();
	dim3 blockSize(16, 16);
	dim3 gridSize((texDesc.Width + blockSize.x - 1) / blockSize.x,
		(texDesc.Height + blockSize.y - 1) / blockSize.y);
	invertColorsKernel<<<gridSize, blockSize>>>(surfacesMap[texture].surface, texDesc.Width, texDesc.Height);

	hipError_t err = hipDeviceSynchronize();
}

void ClearFromMap(ID3D12Resource* texture) {
	surfacesMap[texture].CLEAR();
	surfacesMap.erase(texture);
}

void ClearAllFromMap() {
	for (auto& el : surfacesMap) {
		el.second.CLEAR();
	}
	surfacesMap.clear(); 
}

void InitToMap(ID3D12Resource* texture, ID3D12Device* device) {
	auto iter = surfacesMap.find(texture);
	if (iter!=surfacesMap.end()) {
		PRINTF_FDW("TEXTURE ALREADY IN SURFACE MAP");
		return;
	}

	auto data = CreateSurfaceObject(texture, device);
	if (data.surface) surfacesMap[texture] = data;
}

bool FindInMap(ID3D12Resource* texture) {
	return surfacesMap.find(texture)!=surfacesMap.end();
}
