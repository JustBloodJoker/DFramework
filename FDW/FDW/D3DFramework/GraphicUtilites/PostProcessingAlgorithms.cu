#include "hip/hip_runtime.h"
#include "PostProcessingAlgorithms.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

void CheckCudaErrors(hipError_t error) {
	printf("\nCUDA ERROR --- %d (NULL IT'S OK) \n", error);
}

struct CUDASURFACEDATA {

	hipSurfaceObject_t surface;
	hipMipmappedArray_t cuMipArray;
	hipExternalMemory_t cudaExternalMemory;

	void CLEAR() {
		hipFreeMipmappedArray(cuMipArray);
		hipDestroyExternalMemory(cudaExternalMemory);
		hipDestroySurfaceObject(surface);
	}
};

std::unordered_map<ID3D12Resource*, CUDASURFACEDATA> surfacesMap;


__global__ void InvertColorsKernel(hipSurfaceObject_t surface, unsigned int width, unsigned int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;
	float4 pixel;

	surf2Dread(&pixel, surface, x * sizeof(float4), y);

	pixel.x = 1.0f - pixel.x;
	pixel.y = 1.0f - pixel.y;
	pixel.z = 1.0f - pixel.z;

	surf2Dwrite(pixel, surface, x * sizeof(float4), y );
}

__global__ void GreyColorsKernel(hipSurfaceObject_t surface, unsigned int width, unsigned int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;
	float4 pixel;

	surf2Dread(&pixel, surface, x * sizeof(float4), y);

	float average = (pixel.x + pixel.y + pixel.z) / 3.0;
	pixel = make_float4(average, average, average, pixel.w);

	surf2Dwrite(pixel, surface, x * sizeof(float4), y);
}

__global__ void SharpnessColorsKernel(hipSurfaceObject_t surface, unsigned int width, unsigned int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width-1 || y >= height-1 || x < 1 || y < 1) return;
	

	int kernel[3][3] = { { 0, -1,  0 },
						 { -1,  5, -1 },
						 { 0, -1,  0 } };

	float4 pixel, outputPixel = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	int2 texSize = make_int2(width, height);

	for (int ky = -1; ky <= 1; ky++) {
		for (int kx = -1; kx <= 1; kx++) {
			int2 neighborPos = make_int2(min(max(x + kx, 0), texSize.x - 1),
				min(max(y + ky, 0), texSize.y - 1));

			surf2Dread(&pixel, surface, neighborPos.x * sizeof(float4), neighborPos.y);

			outputPixel.x += kernel[ky + 1][kx + 1] * pixel.x;
			outputPixel.y += kernel[ky + 1][kx + 1] * pixel.y;
			outputPixel.z += kernel[ky + 1][kx + 1] * pixel.z;
		}
	}

	outputPixel.x = min(max(outputPixel.x, 0.0f), 1.0f);
	outputPixel.y = min(max(outputPixel.y, 0.0f), 1.0f);
	outputPixel.z = min(max(outputPixel.z, 0.0f), 1.0f);
	outputPixel.w = 1.0f;

	surf2Dwrite(outputPixel, surface, x * sizeof(float4), y);
}

__global__ void BlurColorKernel(hipSurfaceObject_t surface, unsigned int width, unsigned int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width - 1 || y >= height - 1 || x < 1 || y < 1) return;

	float kernel[3][3] = { { 1.0f / 16, 2.0f / 16, 1.0f / 16 },
						   { 2.0f / 16, 4.0f / 16, 2.0f / 16 },
						   { 1.0f / 16, 2.0f / 16, 1.0f / 16 } };

	float4 pixel, outputPixel = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	int2 texSize = make_int2(width, height);

	for (int ky = -1; ky <= 1; ky++) {
		for (int kx = -1; kx <= 1; kx++) {
			int2 neighborPos = make_int2(min(max(x + kx, 0), texSize.x - 1),
				min(max(y + ky, 0), texSize.y - 1));

			surf2Dread(&pixel, surface, neighborPos.x * sizeof(float4), neighborPos.y);

			outputPixel.x += kernel[ky + 1][kx + 1] * pixel.x;
			outputPixel.y += kernel[ky + 1][kx + 1] * pixel.y;
			outputPixel.z += kernel[ky + 1][kx + 1] * pixel.z;
		}
	}

	outputPixel.x = min(max(outputPixel.x, 0.0f), 1.0f);
	outputPixel.y = min(max(outputPixel.y, 0.0f), 1.0f);
	outputPixel.z = min(max(outputPixel.z, 0.0f), 1.0f);
	outputPixel.w = 1.0f;

	surf2Dwrite(outputPixel, surface, x * sizeof(float4), y);
}

CUDASURFACEDATA CreateSurfaceObject(ID3D12Resource* texture, ID3D12Device* device) {
	CUDASURFACEDATA data{};

	D3D12_RESOURCE_DESC texDesc = texture->GetDesc();
	HANDLE sharedHandle{};
	SECURITY_ATTRIBUTES secAttr{};
	LPCWSTR name{};
	hr = device->CreateSharedHandle(texture, &secAttr, GENERIC_ALL, name, &sharedHandle);
	const auto texAllocInfo = device->GetResourceAllocationInfo(NULL, 1, &texDesc);

	hipExternalMemoryHandleDesc cuExtmemHandleDesc{};
	cuExtmemHandleDesc.type = hipExternalMemoryHandleTypeD3D12Heap;
	cuExtmemHandleDesc.handle.win32.handle = sharedHandle;
	cuExtmemHandleDesc.size = texAllocInfo.SizeInBytes;
	cuExtmemHandleDesc.flags = hipExternalMemoryDedicated;
	CheckCudaErrors(hipImportExternalMemory(&data.cudaExternalMemory, &cuExtmemHandleDesc));

	cudaExternalMemoryMipmappedArrayDesc cuExtmemMipDesc{};
	cuExtmemMipDesc.extent = make_hipExtent(texDesc.Width, texDesc.Height, texDesc.DepthOrArraySize);
	cuExtmemMipDesc.formatDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	cuExtmemMipDesc.numLevels = 1;

	CheckCudaErrors(cudaExternalMemoryGetMappedMipmappedArray(&data.cuMipArray, data.cudaExternalMemory, &cuExtmemMipDesc));

	hipArray_t cuArray{};
	CheckCudaErrors(hipGetMipmappedArrayLevel(&cuArray, data.cuMipArray, 0));

	hipResourceDesc cuResDesc{};
	cuResDesc.resType = hipResourceTypeArray;
	cuResDesc.res.array.array = cuArray;
	CheckCudaErrors(hipCreateSurfaceObject(&data.surface, &cuResDesc));

	return data;
}

void InverseTexture(ID3D12Resource* texture, ID3D12Device* device)
{
	if (!FindInMap(texture)) {
		PRINTF_FDW("CANT FIND TEXTURE SURFACE /// CALL INIT TO MAP");
		InitToMap(texture, device);
	}

	D3D12_RESOURCE_DESC texDesc = texture->GetDesc();
	unsigned width = (unsigned)texDesc.Width;
	unsigned height = (unsigned)texDesc.Height;

	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);
	GreyColorsKernel <<<gridSize, blockSize>>>(surfacesMap[texture].surface, width, height);

	hipError_t err = hipDeviceSynchronize();
}

void GreyEffect(ID3D12Resource* texture, ID3D12Device* device) 
{
	if (!FindInMap(texture)) {
		PRINTF_FDW("CANT FIND TEXTURE SURFACE /// CALL INIT TO MAP");
		InitToMap(texture, device);
	}

	D3D12_RESOURCE_DESC texDesc = texture->GetDesc();
	unsigned width = (unsigned)texDesc.Width;
	unsigned height = (unsigned)texDesc.Height;

	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);
	GreyColorsKernel <<<gridSize, blockSize>>>(surfacesMap[texture].surface, width, height);

	hipError_t err = hipDeviceSynchronize();
}

void SharpnessEffect(ID3D12Resource* texture, ID3D12Device* device)
{
	if (!FindInMap(texture)) {
		PRINTF_FDW("CANT FIND TEXTURE SURFACE /// CALL INIT TO MAP");
		InitToMap(texture, device);
	}

	D3D12_RESOURCE_DESC texDesc = texture->GetDesc();
	unsigned width = (unsigned)texDesc.Width;
	unsigned height = (unsigned)texDesc.Height;

	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);
	SharpnessColorsKernel <<<gridSize, blockSize>>>(surfacesMap[texture].surface, width, height);

	hipError_t err = hipDeviceSynchronize();
}

void BlurEffect(ID3D12Resource* texture, ID3D12Device* device)
{
	if (!FindInMap(texture)) {
		PRINTF_FDW("CANT FIND TEXTURE SURFACE /// CALL INIT TO MAP");
		InitToMap(texture, device);
	}

	D3D12_RESOURCE_DESC texDesc = texture->GetDesc();
	unsigned width = (unsigned)texDesc.Width;
	unsigned height = (unsigned)texDesc.Height;

	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);
	BlurColorKernel <<<gridSize, blockSize>>>(surfacesMap[texture].surface, width, height);

	hipError_t err = hipDeviceSynchronize();
}

void ClearFromMap(ID3D12Resource* texture) {
	surfacesMap[texture].CLEAR();
	surfacesMap.erase(texture);
}

void ClearAllFromMap() {
	for (auto& el : surfacesMap) {
		el.second.CLEAR();
	}
	surfacesMap.clear(); 
}

void InitToMap(ID3D12Resource* texture, ID3D12Device* device) {
	auto iter = surfacesMap.find(texture);
	if (iter!=surfacesMap.end()) {
		PRINTF_FDW("TEXTURE ALREADY IN SURFACE MAP");
		return;
	}

	auto data = CreateSurfaceObject(texture, device);
	if (data.surface) surfacesMap[texture] = data;
}

bool FindInMap(ID3D12Resource* texture) {
	return surfacesMap.find(texture)!=surfacesMap.end();
}
